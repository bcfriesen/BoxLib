
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void init_phi_kernel(double *fab,
                                const int lo1, const int lo2, const int lo3,
                                const int hi1, const int hi2, const int hi3,
                                const double problo1, const double problo2, const double problo3,
                                const double probhi1, const double probhi2, const double probhi3,
                                const int jStride, const int kStride,
                                const int Nghost,
                                const double dx1, const double dx2, const double dx3) {

  int i, j, k, ijk_fab;

  double x, y, z, r2;

  // Convert CUDA thread indices into indicies of the FAB that each thread will
  // modify. Isn't this beautiful??

  i = (blockIdx.x * blockDim.x) + threadIdx.x;
  j = (blockIdx.y * blockDim.y) + threadIdx.y;
  k = (blockIdx.z * blockDim.z) + threadIdx.z;

  if (i >= lo1 && i <= hi1 &&
      j >= lo2 && j <= hi2 &&
      k >= lo3 && k <= hi3) {

    x = problo1 + (double(i)+0.5) * dx1;
    y = problo2 + (double(j)+0.5) * dx2;
    z = problo3 + (double(k)+0.5) * dx3;

    r2 = ((x-0.25)*(x-0.25) + (y-0.25)*(y-0.25) + (z-0.25)*(z-0.25)) * 100.0;

    ijk_fab = (i+Nghost) + (j+Nghost)*jStride + (k+Nghost)*kStride;

    fab[ijk_fab] = 1.0 + std::exp(-r2);
  }

}
